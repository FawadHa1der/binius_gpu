#include <iostream>

#include "common.cuh"

// TODO: In the future we need to move this into enum to return specific errors
bool check_gpu_capabilities() {
	int nDevices;

	CUDA_CHECK(hipGetDeviceCount(&nDevices));

	if (nDevices < 1) {
		std::cerr << "Capabilities Error: There are no cuda capable devices found "
					 "on this machine"
				  << std::endl;
		return false;
	}

	// Assuming the first device is our device
	hipDeviceProp_t prop;
	CUDA_CHECK(hipGetDeviceProperties(&prop, 0));

	if (prop.maxThreadsPerBlock < 1024) {
		std::cerr << "Capabilities Error: less than 1024 threads available per block" << std::endl;
		return false;
	}

	if (prop.sharedMemPerBlock <= (1 << 15)) {
		std::cerr << "Capabilities Error: less than 32kb of shared memory available" << std::endl;
		return false;
	}

	if (prop.maxThreadsDim[0] < 1024 || prop.maxThreadsDim[1] < 1024) {
		std::cerr << "Capabilities Error: less than 1024 threads for x,y dimensions" << std::endl;
		return false;
	}

	if (prop.maxGridSize[0] < (1 << 20) || prop.maxGridSize[1] < (1 << 12) || prop.maxGridSize[2] < (1 << 15)) {
		std::cerr << "Capabilities Error: x,y,z grid dimensions too low" << std::endl;
		return false;
	}

	return true;
}
