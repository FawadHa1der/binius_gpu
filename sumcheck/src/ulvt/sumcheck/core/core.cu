#include "hip/hip_runtime.h"
#include <cstdint>
#include <iostream>

#include "../../finite_fields/circuit_generator/unrolled/binary_tower_unrolled.cuh"
#include "../../utils/bitslicing.cuh"
#include "../utils/constants.hpp"
#include "core.cuh"

__host__ __device__ void evaluate_composition_on_batch_row(
	const uint32_t* first_batch_of_row,
	uint32_t* batch_composition_destination,
	const uint32_t composition_size,
	const uint32_t original_evals_per_col
) {
	memcpy(batch_composition_destination, first_batch_of_row, BITS_WIDTH * sizeof(uint32_t));

	for (int operand_in_composition = 1; operand_in_composition < composition_size; ++operand_in_composition) {
		const uint32_t* nth_batch_of_row =
			first_batch_of_row + operand_in_composition * original_evals_per_col * INTS_PER_VALUE;

		multiply_unrolled<TOWER_HEIGHT>(batch_composition_destination, nth_batch_of_row, batch_composition_destination);
	}
}

__host__ __device__ void fold_batch(
	const uint32_t lower_batch[BITS_WIDTH],
	const uint32_t upper_batch[BITS_WIDTH],
	uint32_t dst_batch[BITS_WIDTH],
	const uint32_t coefficient[BITS_WIDTH],
	const bool is_interpolation
) {
	uint32_t xor_of_halves[BITS_WIDTH];

	for (int i = 0; i < BITS_WIDTH; ++i) {
		xor_of_halves[i] = lower_batch[i] ^ upper_batch[i];
	}

	uint32_t product[BITS_WIDTH];
	memset(product, 0, BITS_WIDTH * sizeof(uint32_t));

	// Multiply chunk-wise based on field height of coefficient
	// For random challenges this will be the full 7
	// For interpolation points this will be no more than 2

	if (is_interpolation) {
		for (int i = 0; i < BITS_WIDTH; i += INTERPOLATION_BITS_WIDTH) {
			multiply_unrolled<INTERPOLATION_TOWER_HEIGHT>(xor_of_halves + i, coefficient, product + i);
		}
	} else {
		multiply_unrolled<TOWER_HEIGHT>(xor_of_halves, coefficient, product);
	}

	for (int i = 0; i < BITS_WIDTH; ++i) {
		dst_batch[i] = lower_batch[i] ^ product[i];
	}
}

void fold_small(
	const uint32_t source[BITS_WIDTH],
	uint32_t destination[BITS_WIDTH],
	const uint32_t coefficient[BITS_WIDTH],
	const uint32_t list_len
) {
	uint32_t half_len = list_len / 2;

	uint32_t batch_to_be_multiplied[BITS_WIDTH];

	memcpy(batch_to_be_multiplied, source, BITS_WIDTH * sizeof(uint32_t));

	for (int i = 0; i < BITS_WIDTH; ++i) {
		batch_to_be_multiplied[i] >>= half_len;  // Move the upper half into the lower half of this operand
		batch_to_be_multiplied[i] ^= source[i];  // Add two halves before multiplying
	}

	uint32_t product[BITS_WIDTH];

	multiply_unrolled<TOWER_HEIGHT>(batch_to_be_multiplied, coefficient, product);

	for (int i = 0; i < BITS_WIDTH; ++i) {
		destination[i] = source[i] ^ product[i];
	}
}

template <> 
__global__ void fold_small_kernel<TOWER_HEIGHT>(uint32_t*  d_src,
                                  uint32_t*        d_dst,
                                  uint32_t*  d_coeff,
                                  uint32_t        list_len,
                                  uint32_t        num_cols)
{
    const uint32_t col = blockIdx.x * blockDim.x + threadIdx.x;
    if (col >= num_cols) return;

    const uint32_t half_len = list_len >> 1;

    /* Pointers to the current column -------------------------------- */
    const uint32_t* src = d_src + col * BITS_WIDTH;
    uint32_t*       dst = d_dst + col * BITS_WIDTH;

    /* Local scratch (lives in registers / local memory) ------------- */
    uint32_t batch[BITS_WIDTH];
#pragma unroll
    for (int i = 0; i < BITS_WIDTH; ++i) {
        uint32_t v = src[i];
        batch[i] = (v >> half_len) ^ v;          // bring upper half down + XOR
    }

    uint32_t prod[BITS_WIDTH];
    multiply_unrolled<TOWER_HEIGHT>(batch, d_coeff, prod);

#pragma unroll
    for (int i = 0; i < BITS_WIDTH; ++i)
        dst[i] = src[i] ^ prod[i];               // final xor into destination
}

// __host__ __device__ void compute_sum(
// 	uint32_t sum[INTS_PER_VALUE],
// 	uint32_t bitsliced_batch[BITS_WIDTH],
// 	const uint32_t num_eval_points_being_summed_unpadded
// ) {
// 	BitsliceUtils<BITS_WIDTH>::bitslice_untranspose(bitsliced_batch);

// 	memset(sum, 0, INTS_PER_VALUE * sizeof(uint32_t));

// 	for (uint32_t i = 0; i < min(BITS_WIDTH, INTS_PER_VALUE * num_eval_points_being_summed_unpadded); ++i) {
// 		sum[i % INTS_PER_VALUE] ^= bitsliced_batch[i];
// 	}
// }

// static inline uint32_t parity32(uint32_t x) {
//     x ^= x >> 16;
//     x ^= x >> 8;
//     x ^= x >> 4;
//     x ^= x >> 2;
//     x ^= x >> 1;
//     return x & 1;
// }

// // without unrolling, potentially faster
// __host__ __device__ void compute_sum(
// 	uint32_t sum[INTS_PER_VALUE],
// 	uint32_t bitsliced_batch[BITS_WIDTH],
// 	const uint32_t       num_eval_points
// ) {
//     /* 0. clear the output ------------------------------------------------ */
//     for (uint32_t lane = 0; lane < INTS_PER_VALUE; ++lane)
//         sum[lane] = 0;

//     /* 1. block anatomy --------------------------------------------------- */
//     const uint32_t full_blocks  = num_eval_points >> 5;        /* N / 32 */
//     const uint32_t tail_points  = num_eval_points & 31u;       /* N % 32 */
//     const uint32_t total_blocks = full_blocks + (tail_points ? 1u : 0u);

//     /* 2. iterate over every bit position and every 32-bit limb ---------- */
//     for (uint32_t bit = 0; bit < 32u; ++bit) {
//         for (uint32_t lane = 0; lane < INTS_PER_VALUE; ++lane) {

//             uint32_t bit_parity = 0;

//             /* ---- full 32-point blocks --------------------------------- */
//             for (uint32_t blk = 0; blk < full_blocks; ++blk) {

// #if BIT_MAJOR_LAYOUT
//                 uint32_t slice =
//                     bitsliced_batch[ blk*BITS_WIDTH + bit*INTS_PER_VALUE + lane ];
// #else
//                 uint32_t slice =
//                     bitsliced_batch[ blk*BITS_WIDTH + lane*32u + bit ];
// #endif
//                 bit_parity ^= parity32(slice);                /* XOR parity */
//             }

//             /* ---- tail block (0 < tail_points < 32) -------------------- */
//             if (tail_points) {

// #if BIT_MAJOR_LAYOUT
//                 uint32_t slice =
//                     bitsliced_batch[ full_blocks*BITS_WIDTH + bit*INTS_PER_VALUE + lane ];
// #else
//                 uint32_t slice =
//                     bitsliced_batch[ full_blocks*BITS_WIDTH + lane*32u + bit ];
// #endif
//                 const uint32_t mask = (1u << tail_points) - 1u;
//                 bit_parity ^= parity32(slice & mask);
//             }

//             /* ---- drop the single parity bit into the result limb ------ */
//             sum[lane] ^= bit_parity << bit;                   /* XOR – safe for reuse */
//         }
//     }
// }

/*****************************************************************************
*  compute_sum.cu  –  CUDA version that uses the native POPC unit            *
*                                                                            *
*  • Handles any value width:   value_bits = 32 × INTS_PER_VALUE             *
*  • Handles any batch size N   (works for N ≤ 32 and N > 32)                *
*  • No atomics, no shared-memory reduction                                  *
*  • Parity is computed with a single POPC instruction per slice             *
*                                                                            *
*  Pick **exactly one** layout flag below to match how your slices are laid   *
*  out in memory.                                                             *
*****************************************************************************/

#define INTS_PER_VALUE    4      // 128-bit value  → 4 × 32-bit limbs
#define BIT_MAJOR_LAYOUT  0      // slice index =  block*W + bit*L + lane
#define LANE_MAJOR_LAYOUT 1      // slice index =  block*W + lane*32 + bit
static_assert(BIT_MAJOR_LAYOUT ^ LANE_MAJOR_LAYOUT, "pick exactly one");

#define BITS_PER_LIMB  32
#define BITS_WIDTH    (BITS_PER_LIMB * INTS_PER_VALUE)

/* --------------------------------------------------------------------- */
/* Kernel: one thread per bit, one block per limb                        */
/* --------------------------------------------------------------------- */
__global__ void compute_sum_kernel(uint32_t      *sum_out,   // OUT: INTS_PER_VALUE words
                                   const uint32_t *slices,   // IN : ≥ ceil(N/32) * BITS_WIDTH words
                                   uint32_t        N)        // IN : evaluation-point count (≥1)
{
    const uint32_t lane   = blockIdx.x;    // which 32-bit limb (0 … INTS_PER_VALUE-1)
    const uint32_t bit    = threadIdx.x;   // which bit position (0 … 31)
    const uint32_t blocks = (N + 31) >> 5; // ceil(N/32)
    const uint32_t tail   =  N & 31u;      // N mod 32   (0 → no tail block)

    uint32_t parity = 0;

    /* ---- all full 32-point blocks ----------------------------------- */
    for (uint32_t blk = 0; blk < blocks - (tail ? 1u : 0u); ++blk) {
#if BIT_MAJOR_LAYOUT
        uint32_t slice = slices[ blk*BITS_WIDTH + bit*INTS_PER_VALUE + lane ];
#else
        uint32_t slice = slices[ blk*BITS_WIDTH + lane*32u + bit ];
#endif
        parity ^= (__popc(slice) & 1u);             // POPC gives 32-bit parity in 1 inst
    }

    /* ---- optional tail block (mask out padding bits) ---------------- */
    if (tail) {
#if BIT_MAJOR_LAYOUT
        uint32_t slice = slices[ (blocks-1u)*BITS_WIDTH + bit*INTS_PER_VALUE + lane ];
#else
        uint32_t slice = slices[ (blocks-1u)*BITS_WIDTH + lane*32u + bit ];
#endif
        slice &= (1u << tail) - 1u;                 // keep only the real N % 32 bits
        parity ^= (__popc(slice) & 1u);
    }

    /* ---- warp-wide pack: thread-bit → result word ------------------- */
    uint32_t limb_word = __ballot_sync(0xFFFFFFFFu, parity);

    /* ---- one thread per block writes its limb ----------------------- */
    if (threadIdx.x == 0)
        sum_out[lane] = limb_word;                  // no atomics – block owns this limb
}

/* --------------------------------------------------------------------- */
/* Convenience launcher (host side)                                      */
/* --------------------------------------------------------------------- */
void compute_sum(
	uint32_t sum[INTS_PER_VALUE],
	uint32_t bitsliced_batch[BITS_WIDTH],
	const uint32_t       num_eval_points)     /* N ≥ 1 */
{
    /* 1. Work geometry ------------------------------------------------- */
    const uint32_t blocks_host = (num_eval_points + 31) >> 5;   /* ceil(N/32) */
    const size_t   slice_words = static_cast<size_t>(blocks_host) * BITS_WIDTH;

    /* 2. Allocate device memory --------------------------------------- */
    uint32_t *d_slices = nullptr, *d_sum = nullptr;
    hipMalloc(&d_slices, slice_words * sizeof(uint32_t));
    hipMalloc(&d_sum,    INTS_PER_VALUE * sizeof(uint32_t));

    /* 3. Copy slices to the device ------------------------------------ */
    hipMemcpy(d_slices, bitsliced_batch,
               slice_words * sizeof(uint32_t), hipMemcpyHostToDevice);

    /* 4. Launch: one 32-thread block per limb ------------------------- */
    dim3 grid (INTS_PER_VALUE);
    dim3 block(32);
    compute_sum_kernel<<<grid, block>>>(d_sum, d_slices, num_eval_points);

    /* 5. Copy the result back ----------------------------------------- */
    hipMemcpy(sum, d_sum,
               INTS_PER_VALUE * sizeof(uint32_t), hipMemcpyDeviceToHost);

    /* 6. Cleanup ------------------------------------------------------- */
    hipFree(d_slices);
    hipFree(d_sum);
}